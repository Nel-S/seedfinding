// Finds the internal states that could displace the player's spawnpoint the farthest in Minecraft Java Beta 1.8(?) - 1.12.
// Must be compiled and linked to 1_idealInternalStatesSplit-MPI.cpp to function.
// (Much of this was taken from https://github.com/KaptainWutax/Kaktoos/blob/master/kaktoos.cu.)

// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif


#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <stdio.h>
#include <mutex>
#include <thread>

#define SEEDS_ARRAY_CAPACITY 1024

const uint64_t GLOBAL_START_STATE = 0;
const uint64_t GLOBAL_STATES_TO_CHECK = 1ULL << 48;
const uint64_t BLOCK_SIZE = 256;
const uint64_t WORK_UNIT_SIZE = 1ULL << 23;
// Best state so far: 1459339358529, (-4, -6357) = 40411465
const uint64_t DIST_THRESHOLD = 37000000;
// const uint64_t DIST_THRESHOLD = 30000000;

// The maximum number of GPUs the system will try to use per cluster node; you shouldn't need to touch this
const int MAX_GPUS_PER_NODE = INT_MAX;

// -----------------------------------------------------------------------
// GPU Helper Functions

// new Random(seed)
__device__ inline void setSeed(uint64_t *random, const uint64_t seed) {
    *random = (seed ^ 25214903917ULL) & ((1ULL << 48) - 1);
}

// Random::nextInt(bound)
__device__ int32_t nextInt(uint64_t *seed, int16_t bound) {
    if ((bound & -bound) == bound) {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        return (int32_t)((bound * (*seed >> 17)) >> 31);
    }
    int32_t bits, value;
    do {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        bits = *seed >> 17;
        value = bits % bound;
    } while (bits - value + (bound - 1) < 0);
    return value;
}
// -------------------------------------------------------------------------
// Main GPU code

typedef struct {
    uint_fast64_t internalState;
    int_fast16_t i;
    uint_fast64_t squaredDist;
} SeedsArray;

typedef struct {
    uint64_t startState, statesToCheck;
    SeedsArray *seedsArray;
    unsigned long long *seedsArraySize;
} GPUnode;

__global__ void test(uint64_t offse, SeedsArray *seedsArray, unsigned long long *seedsArraySize) {
    uint_fast64_t state = blockIdx.x * blockDim.x + threadIdx.x + offse;
    uint_fast64_t random = state;
    uint_fast64_t currentDist, bestDist = 0;
    int_fast16_t posX = 0, posZ = 0, bestI;
    for (int_fast16_t i = 0; i < 1000; ++i) {
        int disp = nextInt(&random, 64);
        posX += disp - nextInt(&random, 64);
        disp = nextInt(&random, 64);
        posZ += disp - nextInt(&random, 64);
        currentDist = posX * posX + posZ * posZ;
        if (currentDist > bestDist) {
            bestDist = currentDist;
            bestI = i;
        }
    }
    if (bestDist >= DIST_THRESHOLD) {
        // printf("%" PRId64"\t%d\t%d\t(%d)\t%" PRIu64 "\n", state, posX, posZ, i, currentDist);
        uint64_t index = atomicAdd(seedsArraySize, 1ull);
        if (index >= SEEDS_ARRAY_CAPACITY)
            return;
        seedsArray[index].internalState = state;
        seedsArray[index].i = bestI;
        seedsArray[index].squaredDist = bestDist;
    }
}
// ------------------------------------------------------------------------------------
// Host code and variables

GPUnode *nodes;
uint64_t startState, statesToCheck, offset = 0;
std::mutex offsetMutex;
#define MIN(X, Y) (X < Y ? X : Y)

// Defined in MPI section
extern void sendMessage(unsigned long long *message);

void launchGPU(int commrank, int index) {
    hipError_t error;
    if (error = hipSetDevice(index)) {
        fprintf(stderr, "Node %d: launchGPU(%d): cudaSetDevice(): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    if (error = hipMallocManaged(&nodes[index].seedsArray, sizeof(*nodes->seedsArray) * SEEDS_ARRAY_CAPACITY)) {
        fprintf(stderr, "Node %d: launchGPU(%d): cudaMallocManaged(seedsArray): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    if (error = hipMallocManaged(&nodes[index].seedsArraySize, sizeof(*nodes->seedsArraySize))) {
        fprintf(stderr, "Node %d: launchGPU(%d): cudaMallocManaged(seedsArraySize): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }

    unsigned long long message[3];
    while (offset < statesToCheck) {
        *nodes[index].seedsArraySize = 0;
        test<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>>(startState + offset, nodes[index].seedsArray, nodes[index].seedsArraySize);
        offsetMutex.lock();
        offset += WORK_UNIT_SIZE;
        offsetMutex.unlock();
        if (error = hipDeviceSynchronize()) {
            fprintf(stderr, "Node %d: launchGPU(%d): cudaDeviceSynchronize(): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
            exit(1);
        }
        for (uint64_t i = 0; i < *nodes[index].seedsArraySize && i < SEEDS_ARRAY_CAPACITY; ++i) {
            message[0] = nodes[index].seedsArray[i].internalState;
            message[1] = nodes[index].seedsArray[i].i;
            message[2] = nodes[index].seedsArray[i].squaredDist;
            sendMessage(message);
        }
    }
    message[0] = message[1] = message[2] = ULLONG_MAX;
    sendMessage(message);
}

extern void launchNode(int commrank, int commsize) {
    hipError_t error;
    int gpuCount;
    if (error = hipGetDeviceCount(&gpuCount)) {
        fprintf(stderr, "Node %d: cudaGetDeviceCount(): %s (%s).\n", commrank, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    if (gpuCount > MAX_GPUS_PER_NODE)
        gpuCount = MAX_GPUS_PER_NODE;
    if (gpuCount <= 0) {
        fprintf(stderr, "Node %d: cudaGetDeviceCount(): Zero or an invalid number of CUDA devices were found (%d).\n", commrank, gpuCount);
        exit(1);
    }
    nodes = (GPUnode *)malloc(gpuCount * sizeof(GPUnode));
    if (!nodes) {
        fprintf(stderr, "Node %d: nodes = malloc(%zd): Could not allocate %zd bytes.\n", commrank, gpuCount * sizeof(GPUnode), gpuCount * sizeof(GPUnode));
        exit(1);
    }
    statesToCheck = GLOBAL_STATES_TO_CHECK / (commsize - 1) + (commrank - 1 < GLOBAL_STATES_TO_CHECK % (commsize - 1));
    startState = GLOBAL_START_STATE + (GLOBAL_STATES_TO_CHECK / (commsize - 1)) * (commrank - 1) + MIN(commrank - 1, GLOBAL_STATES_TO_CHECK % (commsize - 1));

    // std::thread threads[gpuCount];
    std::thread *threads = (std::thread *)malloc(gpuCount * sizeof(std::thread));
    if (!threads) {
        fprintf(stderr, "Node %d: threads = malloc(%zd): Could not allocate %zd bytes.\n", commrank, gpuCount * sizeof(std::thread), gpuCount * sizeof(std::thread));
        exit(1);
    }
    for (int i = 0; i < gpuCount; ++i) threads[i] = std::thread(launchGPU, commrank, i);
    for (int i = 0; i < gpuCount; ++i) threads[i].join();
    free(nodes);
    free(threads);
}