// Finds the internal states that could displace the player's spawnpoint the farthest in Minecraft Java Beta 1.8(?) - 1.12.
// This was almost entirely ripped from https://github.com/KaptainWutax/Kaktoos/blob/master/kaktoos.cu.

// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __CUDACC__
#include <device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif


#include <hip/hip_runtime.h>
#include <inttypes.h>
// #include <memory.h>
#include <stdio.h>
// #include <time.h>
#include <thread>
// #include <vector>
#include <mutex>
#include <chrono>

#define SEEDS_ARRAY_CAPACITY 1024

const uint64_t START_STATE = 0;
const uint64_t STATES_TO_CHECK = 1ULL << 36;
const uint64_t BLOCK_SIZE = 256;
const uint64_t WORK_UNIT_SIZE = 1ULL << 23;
// const uint64_t DIST_THRESHOLD = 40411465; // Best so far: 1459339358529   -4      -6357
// const uint64_t DIST_THRESHOLD = 37000000;
const uint64_t DIST_THRESHOLD = 30000000;
const std::string FILEPATH_PREFIX = "internalStates";

#ifndef GPU_COUNT
#define GPU_COUNT 1
#endif

// -----------------------------------------------------------------------
// new Random(seed)
__device__ inline void setSeed(uint64_t *random, const uint64_t seed) {
    *random = (seed ^ 25214903917ULL) & ((1ULL << 48) - 1);
}

// Random::nextInt(bound)
__device__ int32_t nextInt(uint64_t *seed, int16_t bound) {
    if ((bound & -bound) == bound) {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        return (int32_t)((bound * (*seed >> 17)) >> 31);
    }
    int32_t bits, value;
    do {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        bits = *seed >> 17;
        value = bits % bound;
    } while (bits - value + (bound - 1) < 0);
    return value;
}
// -------------------------------------------------------------------------

typedef struct {
    uint_fast64_t internalState;
    int_fast16_t i;
    uint_fast64_t squaredDist;
} SeedsArray;

typedef struct {
    int GPU;
    SeedsArray *seedsArray;
    uint32_t *seedsArraySize;
} GPUnode;

__global__ void test(uint64_t offse, SeedsArray *seedsArray, uint32_t *seedsArraySize) {
    uint_fast64_t state = START_STATE + blockIdx.x * blockDim.x + threadIdx.x + offse;
    uint_fast64_t random = state;
    uint_fast64_t currentDist, bestDist = 0;
    int_fast16_t posX = 0, posZ = 0, bestI;
    for (int_fast16_t i = 0; i < 1000; ++i) {
        int disp = nextInt(&random, 64);
        posX += disp - nextInt(&random, 64);
        disp = nextInt(&random, 64);
        posZ += disp - nextInt(&random, 64);
        currentDist = posX * posX + posZ * posZ;
        if (currentDist > bestDist) {
            bestDist = currentDist;
            bestI = i;
        }
    }
    if (bestDist >= DIST_THRESHOLD) {
        // printf("%" PRId64"\t%d\t%d\t(%d)\t%" PRIu64 "\n", state, posX, posZ, i, currentDist);
        uint32_t index = atomicAdd(seedsArraySize, 1u);
        if (index >= SEEDS_ARRAY_CAPACITY) return;
        seedsArray[index].internalState = state;
        seedsArray[index].i = bestI;
        seedsArray[index].squaredDist = bestDist;
    }
}
// ------------------------------------------------------------------------------------

GPUnode nodes[GPU_COUNT];
uint64_t offset = 0;
std::mutex offsetMutex; //, fileMutex;
// FILE *file;

void gpu_manager(int index) {
    hipError_t error;
    if (error = hipSetDevice(index)) {
        fprintf(stderr, "gpu_manager(%d): cudaSetDevice(): %s (%s).\n", index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    FILE *file = fopen((FILEPATH_PREFIX + "_" + std::to_string(index) + ".txt").c_str(), "w");
    if (!file) {
        fprintf(stderr, "fopen(): Filepath %s could not be opened.\n", FILEPATH_PREFIX.c_str());
        exit(1);
    }
    nodes[index].GPU = index;
    if (error = hipMallocManaged(&nodes[index].seedsArray, sizeof(*nodes->seedsArray) * SEEDS_ARRAY_CAPACITY)) {
        fprintf(stderr, "gpu_manager(%d): cudaMallocManaged(seedsArray): %s (%s).\n", index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    if (error = hipMallocManaged(&nodes[index].seedsArraySize, sizeof(*nodes->seedsArraySize))) {
        fprintf(stderr, "gpu_manager(%d): cudaMallocManaged(seedsArraySize): %s (%s).\n", index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }

    while (offset < STATES_TO_CHECK) {
        *nodes[index].seedsArraySize = 0;
        test<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>>(offset, nodes[index].seedsArray, nodes[index].seedsArraySize);
        offsetMutex.lock();
        offset += WORK_UNIT_SIZE;
        offsetMutex.unlock();
        if (error = hipDeviceSynchronize()) {
            fprintf(stderr, "gpu_manager(%d): cudaDeviceSynchronize(): %s (%s).\n", index, hipGetErrorString(error), hipGetErrorName(error));
            exit(1);
        }
        // fileMutex.lock();
        for (uint32_t i = 0; i < *nodes[index].seedsArraySize && i < SEEDS_ARRAY_CAPACITY; ++i) {
            fprintf(file, "%" PRIuFAST64 "\t%" PRIdFAST16 "\t%" PRIuFAST64 "\n", nodes[index].seedsArray[i].internalState, nodes[index].seedsArray[i].i, nodes[index].seedsArray[i].squaredDist);
        }
        fflush(file);
        // fileMutex.unlock();
    }
    fclose(file);
}

int main() {
    // file = fopen(FILEPATH_PREFIX.c_str(), "w");
    // if (!file) {
    //     fprintf(stderr, "fopen(): Filepath %s could not be opened.\n", FILEPATH_PREFIX.c_str());
    //     exit(1);
    // }

    std::thread threads[GPU_COUNT];
    // time_t startTime = time(NULL), currentTime;
    for (int i = 0; i < GPU_COUNT; i++) threads[i] = std::thread(gpu_manager, i);
    for (int i = 0; i < GPU_COUNT; i++) threads[i].join();

    // using namespace std::chrono_literals;
    // while (offset < STATES_TO_CHECK) {
    // //     time(&currentTime);
    // //     int timeElapsed = (int)(currentTime - startTime);
    // //     double speed = (double)(offset) / (double)timeElapsed / 1000000.0;
    // //     printf("Searched %lld seeds, offset: %lld. Time elapsed: %ds. Speed: %.2fm seeds/s. %f%%\n", (long long int)(offset), (long long int)offset, timeElapsed, speed, (double)offset / SEEDS_TO_CHECK * 100);
    //     std::this_thread::sleep_for(3s);
    // }

    // fclose(file);
    // time(&currentTime);
    // printf("(%d seconds)\n", (int)(currentTime - startTime));
    return 0;
}