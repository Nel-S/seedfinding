#include "hip/hip_runtime.h"
// Finds the internal states that could displace the player's spawnpoint the farthest in Minecraft Java Beta 1.8(?) - 1.12.

#include <hip/hip_runtime.h>
#include <cinttypes>
#include <cstdio>

constexpr const uint64_t START_STATE = 285936611421756;
constexpr const uint64_t STATES_TO_CHECK = (1ULL << 48) - START_STATE;
constexpr const uint64_t BLOCK_SIZE = 256;
constexpr const uint64_t WORK_UNIT_SIZE = 1ULL << 32;
// const uint64_t DIST_THRESHOLD = 40411465; // Best so far: 1459339358529   -4      -6357
// const uint64_t DIST_THRESHOLD = 37000000;
constexpr const uint64_t DIST_THRESHOLD = 36178138;
constexpr const char *FILEPATH = "internalStates (36178138).txt";

// -----------------------------------------------------------------------
// new Random(seed)
__device__ inline void setSeed(uint64_t *random, const uint64_t seed) {
    *random = (seed ^ 25214903917ULL) & ((1ULL << 48) - 1);
}

// Random::nextInt(bound)
__device__ int32_t nextInt(uint64_t *seed, int16_t bound) {
    if ((bound & -bound) == bound) {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        return (int32_t)((bound * (*seed >> 17)) >> 31);
    }
    int32_t bits, value;
    do {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        bits = *seed >> 17;
        value = bits % bound;
    } while (bits - value + (bound - 1) < 0);
    return value;
}
// -------------------------------------------------------------------------

typedef struct {
    uint_fast64_t internalState;
    int_fast16_t i;
    uint_fast64_t squaredDist;
} SeedsArray;

constexpr uint64_t SEEDS_ARRAY_CAPACITY = 8192;
__managed__ SeedsArray seedsArray[SEEDS_ARRAY_CAPACITY];
__managed__ uint64_t seedsArraySize;

__global__ void test(uint64_t offse) {
    uint_fast64_t state = START_STATE + blockIdx.x * blockDim.x + threadIdx.x + offse;
    uint_fast64_t random = state;
    uint_fast64_t currentDist, bestDist = 0;
    int_fast16_t posX = 0, posZ = 0, bestI;
    for (int_fast16_t i = 0; i < 1000; ++i) {
        int disp = nextInt(&random, 64);
        posX += disp - nextInt(&random, 64);
        disp = nextInt(&random, 64);
        posZ += disp - nextInt(&random, 64);
        currentDist = posX * posX + posZ * posZ;
        if (currentDist > bestDist) {
            bestDist = currentDist;
            bestI = i;
        }
    }
    if (bestDist < DIST_THRESHOLD) return;
    // printf("%" PRId64"\t%d\t%d\t(%d)\t%" PRIu64 "\n", state, posX, posZ, i, currentDist);
    uint64_t index = atomicAdd(&seedsArraySize, 1);
    if (index >= SEEDS_ARRAY_CAPACITY) return;
    seedsArray[index] = {state, bestI, bestDist};
}
// ------------------------------------------------------------------------------------

template <class T>
constexpr T& min(const T &first, const T &second) {
    return first < second ? first : second;
}

int main() {
    hipError_t error;
    if (error = hipSetDevice(0)) {
        fprintf(stderr, "hipSetDevice(): %s (%s).\n", hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    FILE *file = fopen(FILEPATH, "w");
    if (!file) {
        fprintf(stderr, "fopen(): Filepath %s could not be opened.\n", FILEPATH);
        exit(1);
    }

    for (uint64_t offset = 0; offset < STATES_TO_CHECK; offset += WORK_UNIT_SIZE) {
        seedsArraySize = 0;
        test<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE>>>(offset);
        if (error = hipDeviceSynchronize()) {
            fprintf(stderr, "hipDeviceSynchronize(): %s (%s).\n", hipGetErrorString(error), hipGetErrorName(error));
            exit(1);
        }
        // fileMutex.lock();
        for (uint64_t i = 0; i < min(seedsArraySize, SEEDS_ARRAY_CAPACITY); ++i) {
            printf("%" PRIuFAST64 "\t%" PRIdFAST16 "\t%" PRIuFAST64 "\n", seedsArray[i].internalState, seedsArray[i].i, seedsArray[i].squaredDist);
            fprintf(file, "%" PRIuFAST64 "\t%" PRIdFAST16 "\t%" PRIuFAST64 "\n", seedsArray[i].internalState, seedsArray[i].i, seedsArray[i].squaredDist);
        }
        fflush(stdout);
        fflush(file);
        // fileMutex.unlock();
    }
    fclose(file);
}