#include "hip/hip_runtime.h"
// Finds the internal java.util.Random states that could displace the player's spawnpoint the farthest in Minecraft Java Beta 1.8(?) - 1.12.
// (Much of this was taken from https://github.com/KaptainWutax/Kaktoos/blob/master/kaktoos.cu.)

// IDE indexing
#ifdef __JETBRAINS_IDE__
#define __host__
#define __device__
#define __shared__
#define __constant__
#define __global__
#define __HIPCC__
#include <hip/device_functions.h>
#include <__clang_cuda_builtin_vars.h>
#include <__clang_cuda_intrinsics.h>
#include <__clang_cuda_math_forward_declares.h>
#include <__clang_cuda_complex_builtins.h>
#include <__clang_cuda_cmath.h>
#endif

#include <inttypes.h>
#include <mpi.h>
#include <stdio.h>
// #include <chrono>
#include <mutex>
#include <thread>

#define SEEDS_ARRAY_CAPACITY 1024

const uint64_t GLOBAL_START_STATE = 0;
const uint64_t GLOBAL_STATES_TO_CHECK = 1ULL << 36;
const uint64_t BLOCK_SIZE = 256;
const uint64_t WORK_UNIT_SIZE = 1ULL << 23;
// Best state so far: 1459339358529, (-4, -6357) = 40411465
// const uint64_t DIST_THRESHOLD = 37000000;
const uint64_t DIST_THRESHOLD = 30000000;
const std::string FILEPATH_PREFIX = "internalStates";

// const size_t NODES_COUNT = 1;
// const size_t GPUS_PER_NODE = 1;
// The maximum number of GPUs the system will try to use per cluster node; you shouldn't need to touch this
const int MAX_GPUS_PER_NODE = INT_MAX;

// -----------------------------------------------------------------------
// GPU Helper Functions

// new Random(seed)
__device__ inline void setSeed(uint64_t *random, const uint64_t seed) {
    *random = (seed ^ 25214903917ULL) & ((1ULL << 48) - 1);
}

// Random::nextInt(bound)
__device__ int32_t nextInt(uint64_t *seed, int16_t bound) {
    if ((bound & -bound) == bound) {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        return (int32_t)((bound * (*seed >> 17)) >> 31);
    }
    int32_t bits, value;
    do {
        *seed = (*seed * 25214903917ULL + 11ULL) & ((1ULL << 48) - 1);
        bits = *seed >> 17;
        value = bits % bound;
    } while (bits - value + (bound - 1) < 0);
    return value;
}
// -------------------------------------------------------------------------
// Main GPU code

typedef struct {
    uint_fast64_t internalState;
    int_fast16_t i;
    uint_fast64_t squaredDist;
} SeedsArray;
enum {messageInternalState, messageI, messageSquaredDist, messageSize};

typedef struct {
    uint64_t startState, statesToCheck;
    SeedsArray *seedsArray;
    unsigned long long *seedsArraySize;
} GPUnode;

__global__ void test(uint64_t offse, SeedsArray *seedsArray, unsigned long long *seedsArraySize) {
    uint_fast64_t state = blockIdx.x * blockDim.x + threadIdx.x + offse;
    uint_fast64_t random = state;
    uint_fast64_t currentDist, bestDist = 0;
    int_fast16_t posX = 0, posZ = 0, bestI;
    for (int_fast16_t i = 0; i < 1000; ++i) {
        int disp = nextInt(&random, 64);
        posX += disp - nextInt(&random, 64);
        disp = nextInt(&random, 64);
        posZ += disp - nextInt(&random, 64);
        currentDist = posX * posX + posZ * posZ;
        if (currentDist > bestDist) {
            bestDist = currentDist;
            bestI = i;
        }
    }
    if (bestDist >= DIST_THRESHOLD) {
        // printf("%" PRId64"\t%d\t%d\t(%d)\t%" PRIu64 "\n", state, posX, posZ, i, currentDist);
        uint64_t index = atomicAdd(seedsArraySize, 1ull);
        if (index >= SEEDS_ARRAY_CAPACITY) return;
        seedsArray[index].internalState = state;
        seedsArray[index].i = bestI;
        seedsArray[index].squaredDist = bestDist;
    }
}
// ------------------------------------------------------------------------------------
// Host code and variables

int commrank, commsize;
// GPUnode nodes[GPUS_PER_NODE];
GPUnode *nodes;
uint64_t startState = GLOBAL_START_STATE, statesToCheck = GLOBAL_STATES_TO_CHECK, offset = 0;
std::mutex offsetMutex;
#define MIN(X,Y) (X < Y ? X : Y)

void gpu_manager(int index) {
    hipError_t error;
    if (error = hipSetDevice(index)) {
        fprintf(stderr, "Node %d: gpu_manager(%d): hipSetDevice(): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    if (error = hipMallocManaged(&nodes[index].seedsArray, sizeof(*nodes->seedsArray) * SEEDS_ARRAY_CAPACITY)) {
        fprintf(stderr, "Node %d: gpu_manager(%d): hipMallocManaged(seedsArray): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }
    if (error = hipMallocManaged(&nodes[index].seedsArraySize, sizeof(*nodes->seedsArraySize))) {
        fprintf(stderr, "Node %d: gpu_manager(%d): hipMallocManaged(seedsArraySize): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
        exit(1);
    }

    unsigned long long message[messageSize];
    while (offset < statesToCheck) {
        *nodes[index].seedsArraySize = 0;
        test<<<WORK_UNIT_SIZE / BLOCK_SIZE, BLOCK_SIZE, 0>>>(startState + offset, nodes[index].seedsArray, nodes[index].seedsArraySize);
        offsetMutex.lock();
        offset += WORK_UNIT_SIZE;
        offsetMutex.unlock();
        if (error = hipDeviceSynchronize()) {
            fprintf(stderr, "Node %d: gpu_manager(%d): hipDeviceSynchronize(): %s (%s).\n", commrank, index, hipGetErrorString(error), hipGetErrorName(error));
            exit(1);
        }
        for (uint64_t i = 0; i < *nodes[index].seedsArraySize && i < SEEDS_ARRAY_CAPACITY; ++i) {
            message[messageInternalState] = nodes[index].seedsArray[i].internalState;
            message[messageI] = nodes[index].seedsArray[i].i;
            message[messageSquaredDist] = nodes[index].seedsArray[i].squaredDist;
            MPI_Send(message, messageSize, MPI_UNSIGNED_LONG_LONG, 0, 0, MPI_COMM_WORLD);
        }
    }
    message[messageInternalState] = message[messageI] = message[messageSquaredDist] = ULLONG_MAX;
    MPI_Send(message, messageSize, MPI_UNSIGNED_LONG_LONG, 0, 0, MPI_COMM_WORLD);
}

int main() {
    MPI_Init(NULL, NULL);
    MPI_Comm_size(MPI_COMM_WORLD, &commsize);
    MPI_Comm_rank(MPI_COMM_WORLD, &commrank);
    if (commsize < 2) {
        fprintf(stderr, "Node %d: MPI_Comm_size(): commsize = %d must be 2 or greater (1 writer process, 1+ calculation processes).\n", commrank, commsize);
        exit(1);
    }

    if (commrank) {
        hipError_t error;
        int gpuCount;
        if (error = hipGetDeviceCount(&gpuCount)) {
            fprintf(stderr, "Node %d: hipGetDeviceCount(): %s (%s).\n", commrank, hipGetErrorString(error), hipGetErrorName(error));
            exit(1);
        }
        if (gpuCount > MAX_GPUS_PER_NODE) gpuCount = MAX_GPUS_PER_NODE;
        if (gpuCount <= 0) {
            fprintf(stderr, "Node %d: hipGetDeviceCount(): Zero or an invalid number of CUDA devices were found (%d).\n", commrank, gpuCount);
            exit(1);
        }
        nodes = (GPUnode *)malloc(gpuCount*sizeof(GPUnode));
        if (!nodes) {
            fprintf(stderr, "Node %d: nodes = malloc(%zd): Could not allocate %zd bytes.\n", commrank, gpuCount*sizeof(GPUnode), gpuCount*sizeof(GPUnode));
            exit(1);
        }
        statesToCheck = GLOBAL_STATES_TO_CHECK/(commsize - 1) + (commrank - 1 < GLOBAL_STATES_TO_CHECK % (commsize - 1));
        startState = GLOBAL_START_STATE + (GLOBAL_STATES_TO_CHECK/(commsize - 1))*(commrank - 1) + MIN(commrank - 1, GLOBAL_STATES_TO_CHECK % (commsize - 1));
        
        std::thread threads[gpuCount];
        for (int i = 0; i < gpuCount; ++i) threads[i] = std::thread(gpu_manager, i);
        for (int i = 0; i < gpuCount; ++i) threads[i].join();
        free(nodes);
    } else {
        FILE *file = fopen((FILEPATH_PREFIX + ".txt").c_str(), "w");
        if (!file) {
            fprintf(stderr, "Device %d: fopen(\"%s\", \"w\"): Filepath could not be opened.\n", commrank, (FILEPATH_PREFIX + ".txt").c_str());
            exit(1);
        }
        int openConnections = commsize - 1;
        unsigned long long message[messageSize];
        while (openConnections) {
            MPI_Recv(message, messageSize, MPI_UNSIGNED_LONG_LONG, MPI_ANY_SOURCE, MPI_ANY_TAG, MPI_COMM_WORLD, MPI_STATUS_IGNORE);
            if (message[messageInternalState] == ULLONG_MAX && message[messageI] == ULLONG_MAX && message[messageSquaredDist] == ULLONG_MAX) --openConnections;
            else fprintf(file, "%llu\t%llu\t%llu\n", message[messageInternalState], message[messageI], message[messageSquaredDist]);
        }
        fflush(file);
        fclose(file);
    }
    MPI_Finalize();
    return 0;
}